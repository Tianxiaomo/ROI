// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace roi {
int get_cudart_version() {
  return CUDART_VERSION;
}
} // namespace roi
